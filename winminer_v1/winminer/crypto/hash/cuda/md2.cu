#include "hip/hip_runtime.h"
/*
 * md2.cu CUDA Implementation of MD2 digest
 *
 * Copyright (c) 2019 by Adequate Systems, LLC.  All Rights Reserved.
 * See LICENSE.PDF   **** NO WARRANTY ****
 *
 * Date: 12 June 2019
 * Revision: 1
 *
 * This file is subject to the license as found in LICENSE.PDF
 *
 * Based on the public domain Reference Implementation in C, by
 * Brad Conte, original code here:
 *
 * https://github.com/B-Con/crypto-algorithms
 *
 */

/*************************** HEADER FILES ***************************/
#include <stdlib.h>
#include <memory.h>
#include <>
#include <hip/hip_runtime.h>
extern "C" {
#include "md2.cuh"
}
#define MD2_BLOCK_SIZE 16
/**************************** STRUCT ********************************/
typedef struct {
	BYTE data[16];
	BYTE state[48];
	BYTE checksum[16];
	int len;
} CUDA_MD2_CTX;

/**************************** VARIABLES *****************************/
__constant__ BYTE s[256] = {
	41, 46, 67, 201, 162, 216, 124, 1, 61, 54, 84, 161, 236, 240, 6,
	19, 98, 167, 5, 243, 192, 199, 115, 140, 152, 147, 43, 217, 188,
	76, 130, 202, 30, 155, 87, 60, 253, 212, 224, 22, 103, 66, 111, 24,
	138, 23, 229, 18, 190, 78, 196, 214, 218, 158, 222, 73, 160, 251,
	245, 142, 187, 47, 238, 122, 169, 104, 121, 145, 21, 178, 7, 63,
	148, 194, 16, 137, 11, 34, 95, 33, 128, 127, 93, 154, 90, 144, 50,
	39, 53, 62, 204, 231, 191, 247, 151, 3, 255, 25, 48, 179, 72, 165,
	181, 209, 215, 94, 146, 42, 172, 86, 170, 198, 79, 184, 56, 210,
	150, 164, 125, 182, 118, 252, 107, 226, 156, 116, 4, 241, 69, 157,
	112, 89, 100, 113, 135, 32, 134, 91, 207, 101, 230, 45, 168, 2, 27,
	96, 37, 173, 174, 176, 185, 246, 28, 70, 97, 105, 52, 64, 126, 15,
	85, 71, 163, 35, 221, 81, 175, 58, 195, 92, 249, 206, 186, 197,
	234, 38, 44, 83, 13, 110, 133, 40, 132, 9, 211, 223, 205, 244, 65,
	129, 77, 82, 106, 220, 55, 200, 108, 193, 171, 250, 36, 225, 123,
	8, 12, 189, 177, 74, 120, 136, 149, 139, 227, 99, 232, 109, 233,
	203, 213, 254, 59, 0, 29, 57, 242, 239, 183, 14, 102, 88, 208, 228,
	166, 119, 114, 248, 235, 117, 75, 10, 49, 68, 80, 180, 143, 237,
	31, 26, 219, 153, 141, 51, 159, 17, 131, 20
};

/*********************** FUNCTION DEFINITIONS ***********************/
__device__ void cuda_md2_transform(CUDA_MD2_CTX *ctx, BYTE data[])
{
	int j,k,t;

	//memcpy(&ctx->state[16], data);
	for (j=0; j < 16; ++j) {
		ctx->state[j + 16] = data[j];
		ctx->state[j + 32] = (ctx->state[j+16] ^ ctx->state[j]);
	}

	t = 0;
	for (j = 0; j < 18; ++j) {
		for (k = 0; k < 48; ++k) {
			ctx->state[k] ^= s[t];
			t = ctx->state[k];
		}
		t = (t+j) & 0xFF;
	}

	t = ctx->checksum[15];
	for (j=0; j < 16; ++j) {
		ctx->checksum[j] ^= s[data[j] ^ t];
		t = ctx->checksum[j];
	}
}

__device__ void cuda_md2_init(CUDA_MD2_CTX *ctx)
{
	int i;

	for (i=0; i < 48; ++i)
		ctx->state[i] = 0;
	for (i=0; i < 16; ++i)
		ctx->checksum[i] = 0;
	ctx->len = 0;
}

__device__ void cuda_md2_update(CUDA_MD2_CTX *ctx, const BYTE data[], size_t len)
{
	size_t i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->len] = data[i];
		ctx->len++;
		if (ctx->len == MD2_BLOCK_SIZE) {
			cuda_md2_transform(ctx, ctx->data);
			ctx->len = 0;
		}
	}
}

__device__ void cuda_md2_final(CUDA_MD2_CTX *ctx, BYTE hash[])
{
	int to_pad;

	to_pad = MD2_BLOCK_SIZE - ctx->len;

	while (ctx->len < MD2_BLOCK_SIZE)
		ctx->data[ctx->len++] = to_pad;

	cuda_md2_transform(ctx, ctx->data);
	cuda_md2_transform(ctx, ctx->checksum);

	memcpy(hash, ctx->state, MD2_BLOCK_SIZE);
}

__global__ void kernel_md2_hash(BYTE* indata, WORD inlen, BYTE* outdata, WORD n_batch)
{
	WORD thread = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread >= n_batch)
	{
		return;
	}
	BYTE* in = indata  + thread * inlen;
	BYTE* out = outdata  + thread * MD2_BLOCK_SIZE;
	CUDA_MD2_CTX ctx;
	cuda_md2_init(&ctx);
	cuda_md2_update(&ctx, in, inlen);
	cuda_md2_final(&ctx, out);
}
extern "C" {
void mcm_cuda_md2_hash_batch(BYTE *in, WORD inlen, BYTE *out, WORD n_batch) {
	BYTE *cuda_indata;
	BYTE *cuda_outdata;
	hipMalloc(&cuda_indata, inlen * n_batch);
	hipMalloc(&cuda_outdata, MD2_BLOCK_SIZE * n_batch);
	hipMemcpy(cuda_indata, in, inlen * n_batch, hipMemcpyHostToDevice);

	WORD thread = 256;
	WORD block = (n_batch + thread - 1) / thread;

	kernel_md2_hash << < block, thread >> > (cuda_indata, inlen, cuda_outdata, n_batch);
	hipMemcpy(out, cuda_outdata, MD2_BLOCK_SIZE * n_batch, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error cuda md2 hash: %s \n", hipGetErrorString(error));
	}
	hipFree(cuda_indata);
	hipFree(cuda_outdata);
}
}
