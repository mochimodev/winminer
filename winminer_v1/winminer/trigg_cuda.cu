#include "hip/hip_runtime.h"
/* trigg_cuda.cu CUDA Implementation of Trigg's Algorithm
 *
 * Copyright (c) 2019 by Adequate Systems, LLC.  All Rights Reserved.
 * See LICENSE.PDF   **** NO WARRANTY ****
 * https://github.com/mochimodev/mochimo/raw/master/LICENSE.PDF
 *
 * Date: 26 January 2019
 *
*/

#include "winminer.h"
#include "trigg_cuda.h"

/* Lines 19 through 208 of this file were provided by a third party and are not subject to copyright
 * or ownership claims by Adequate Systems.  These lines represent a CUDA implementation of SHA-256
 * that an anonymous community member wrote for us.  Please note sha256 is based on public domain code
 * by Brad Conte (brad AT bradconte.com).
 * https://raw.githubusercontent.com/B-Con/crypto-algorithms/master/sha256.c
*/

__constant__ static uint32_t __align__(8) c_midstate256[8];
__constant__ static uint32_t __align__(8) c_input32[8];
__constant__ static uint32_t __align__(8) c_blockNumber8[2];
__constant__ static uint8_t __align__(8) c_difficulty;
__constant__ static uint32_t __align__(8) c_K[64] =
{
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

#ifdef __CUDA_ARCH__
__device__ __forceinline__ uint32_t cuda_swab32(uint32_t x)
{
	/* device */
	return __byte_perm(x, x, 0x0123);
}
#else
/* host */
#define cuda_swab32(x) \
    ((((x) << 24) & 0xff000000u) | (((x) << 8) & 0x00ff0000u) | \
        (((x) >> 8) & 0x0000ff00u) | (((x) >> 24) & 0x000000ffu))
#endif

#define xor3b(a,b,c) (a ^ b ^ c)
#define ROTR32(x, n) __funnelshift_r( (x), (x), (n) )

__device__ __forceinline__ uint64_t xandx(uint64_t a, uint64_t b, uint64_t c)
{
	uint64_t result;
	asm("{ // xandx \n\t"
		".reg .u64 n;\n\t"
		"xor.b64 %0, %2, %3;\n\t"
		"and.b64 n, %0, %1;\n\t"
		"xor.b64 %0, n, %3;\n\t"
		"}\n" : "=l"(result) : "l"(a), "l"(b), "l"(c));
	return result;
}

#define AS_UINT2(addr) *((uint2*)(addr))

__device__ __forceinline__ uint32_t bsg2_0(const uint32_t x)
{
	return xor3b(ROTR32(x, 2), ROTR32(x, 13), ROTR32(x, 22));
}

__device__ __forceinline__ uint32_t bsg2_1(const uint32_t x)
{
	return xor3b(ROTR32(x, 6), ROTR32(x, 11), ROTR32(x, 25));
}

__device__ __forceinline__ uint32_t ssg2_0(const uint32_t x)
{
	return xor3b(ROTR32(x, 7), ROTR32(x, 18), (x >> 3));
}

__device__ __forceinline__ uint32_t ssg2_1(const uint32_t x)
{
	return xor3b(ROTR32(x, 17), ROTR32(x, 19), (x >> 10));
}

__device__ __forceinline__ uint32_t andor32(const uint32_t a, const uint32_t b, const uint32_t c)
{
	uint32_t result;
	asm("{\n\t"
		".reg .u32 m,n,o;\n\t"
		"and.b32 m,  %1, %2;\n\t"
		" or.b32 n,  %1, %2;\n\t"
		"and.b32 o,   n, %3;\n\t"
		" or.b32 %0,  m, o ;\n\t"
		"}\n\t" : "=r"(result) : "r"(a), "r"(b), "r"(c)
	);
	return result;
}

__device__ __forceinline__ uint2 vectorizeswap(uint64_t v)
{
	uint2 result;
	asm("mov.b64 {%0,%1},%2; \n\t"
		: "=r"(result.y), "=r"(result.x) : "l"(v));
	return result;
}

__device__ static void sha2_step1(uint32_t a, uint32_t b, uint32_t c, uint32_t &d, uint32_t e, uint32_t f, uint32_t g, uint32_t &h,
	uint32_t in, const uint32_t Kshared)
{
	uint32_t t1, t2;
	uint32_t vxandx = xandx(e, f, g);
	uint32_t bsg21 = bsg2_1(e);
	uint32_t bsg20 = bsg2_0(a);
	uint32_t andorv = andor32(a, b, c);

	t1 = h + bsg21 + vxandx + Kshared + in;
	t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

__device__ static void sha2_step2(uint32_t a, uint32_t b, uint32_t c, uint32_t &d, uint32_t e, uint32_t f, uint32_t g, uint32_t &h,
	uint32_t* in, uint32_t pc, const uint32_t Kshared)
{
	uint32_t t1, t2;

	int pcidx1 = (pc - 2) & 0xF;
	int pcidx2 = (pc - 7) & 0xF;
	int pcidx3 = (pc - 15) & 0xF;

	uint32_t inx0 = in[pc];
	uint32_t inx1 = in[pcidx1];
	uint32_t inx2 = in[pcidx2];
	uint32_t inx3 = in[pcidx3];

	uint32_t ssg21 = ssg2_1(inx1);
	uint32_t ssg20 = ssg2_0(inx3);
	uint32_t vxandx = xandx(e, f, g);
	uint32_t bsg21 = bsg2_1(e);
	uint32_t bsg20 = bsg2_0(a);
	uint32_t andorv = andor32(a, b, c);

	in[pc] = ssg21 + inx2 + ssg20 + inx0;

	t1 = h + bsg21 + vxandx + Kshared + in[pc];
	t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

__device__ static void sha256_round(uint32_t* in, uint32_t* state, uint32_t* const Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha2_step1(a, b, c, d, e, f, g, h, in[0], Kshared[0]);
	sha2_step1(h, a, b, c, d, e, f, g, in[1], Kshared[1]);
	sha2_step1(g, h, a, b, c, d, e, f, in[2], Kshared[2]);
	sha2_step1(f, g, h, a, b, c, d, e, in[3], Kshared[3]);
	sha2_step1(e, f, g, h, a, b, c, d, in[4], Kshared[4]);
	sha2_step1(d, e, f, g, h, a, b, c, in[5], Kshared[5]);
	sha2_step1(c, d, e, f, g, h, a, b, in[6], Kshared[6]);
	sha2_step1(b, c, d, e, f, g, h, a, in[7], Kshared[7]);
	sha2_step1(a, b, c, d, e, f, g, h, in[8], Kshared[8]);
	sha2_step1(h, a, b, c, d, e, f, g, in[9], Kshared[9]);
	sha2_step1(g, h, a, b, c, d, e, f, in[10], Kshared[10]);
	sha2_step1(f, g, h, a, b, c, d, e, in[11], Kshared[11]);
	sha2_step1(e, f, g, h, a, b, c, d, in[12], Kshared[12]);
	sha2_step1(d, e, f, g, h, a, b, c, in[13], Kshared[13]);
	sha2_step1(c, d, e, f, g, h, a, b, in[14], Kshared[14]);
	sha2_step1(b, c, d, e, f, g, h, a, in[15], Kshared[15]);

#pragma unroll
	for (int i = 0; i < 3; i++)
	{
		sha2_step2(a, b, c, d, e, f, g, h, in, 0, Kshared[16 + 16 * i]);
		sha2_step2(h, a, b, c, d, e, f, g, in, 1, Kshared[17 + 16 * i]);
		sha2_step2(g, h, a, b, c, d, e, f, in, 2, Kshared[18 + 16 * i]);
		sha2_step2(f, g, h, a, b, c, d, e, in, 3, Kshared[19 + 16 * i]);
		sha2_step2(e, f, g, h, a, b, c, d, in, 4, Kshared[20 + 16 * i]);
		sha2_step2(d, e, f, g, h, a, b, c, in, 5, Kshared[21 + 16 * i]);
		sha2_step2(c, d, e, f, g, h, a, b, in, 6, Kshared[22 + 16 * i]);
		sha2_step2(b, c, d, e, f, g, h, a, in, 7, Kshared[23 + 16 * i]);
		sha2_step2(a, b, c, d, e, f, g, h, in, 8, Kshared[24 + 16 * i]);
		sha2_step2(h, a, b, c, d, e, f, g, in, 9, Kshared[25 + 16 * i]);
		sha2_step2(g, h, a, b, c, d, e, f, in, 10, Kshared[26 + 16 * i]);
		sha2_step2(f, g, h, a, b, c, d, e, in, 11, Kshared[27 + 16 * i]);
		sha2_step2(e, f, g, h, a, b, c, d, in, 12, Kshared[28 + 16 * i]);
		sha2_step2(d, e, f, g, h, a, b, c, in, 13, Kshared[29 + 16 * i]);
		sha2_step2(c, d, e, f, g, h, a, b, in, 14, Kshared[30 + 16 * i]);
		sha2_step2(b, c, d, e, f, g, h, a, in, 15, Kshared[31 + 16 * i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}
/* Code below this line is property of Adequate Systems, LLC. Copyright 2019. All Rights Reserved.  Please see LICENSE.PDF 
 * for specific license details */
__device__ int gpu_trigg_eval(uint32_t *h, uint8_t d)
{
	uint32_t *bp, n;
	for (bp = h, n = d >> 5; n; n--) {
		if (*bp++ != 0) return 0;
	}
	return __clz(*bp) >= (d & 31);
}

__constant__ static int Z_PREP[8] = { 12,13,14,15,16,17,12,13 }; /* Confirmed */
__constant__ static int Z_ING[32] = { 18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33,34,35,36,37,38,39,40,41,42,43,23,24,31,32,33,34 }; /* Confirmed */
__constant__ static int Z_INF[16] = { 44,45,46,47,48,50,51,52,53,54,55,56,57,58,59,60 }; /* Confirmed */
__constant__ static int Z_ADJ[64] = { 61,62,63,64,65,66,67,68,69,70,71,72,73,74,75,76,77,78,79,80,81,82,83,84,85,86,87,88,89,90,91,92,94,95,96,97,98,99,100,101,102,103,104,105,107,108,109,110,112,114,115,116,117,118,119,120,121,122,123,124,125,126,127,128 }; /* Confirmed */
__constant__ static int Z_AMB[16] = { 77,94,95,96,126,214,217,218,220,222,223,224,225,226,227,228 }; /* Confirmed */
__constant__ static int Z_TIMED[8] = { 84,243,249,250,251,252,253,255 }; /* Confirmed */
__constant__ static int Z_NS[64] = { 129,130,131,132,133,134,135,136,137,138,145,149,154,155,156,157,177,178,179,180,182,183,184,185,186,187,188,189,190,191,192,193,194,196,197,198,199,200,201,202,203,204,205,206,207,208,209,210,211,212,213,241,244,245,246,247,248,249,250,251,252,253,254,255 }; /* Confirmed */
__constant__ static int Z_NPL[32] = { 139,140,141,142,143,144,146,147,148,150,151,153,158,159,160,161,162,163,164,165,166,167,168,169,170,171,172,173,174,175,176,181 }; /* Confirmed */
__constant__ static int Z_MASS[32] = { 214,215,216,217,218,219,220,221,222,223,224,225,226,227,228,229,230,231,232,233,234,235,236,237,238,239,240,242,214,215,216,219 }; /* Confirmed */
__constant__ static int Z_INGINF[32] = { 18,19,20,21,22,25,26,27,28,29,30,36,37,38,39,40,41,42,44,46,47,48,49,51,52,53,54,55,56,57,58,59 }; /* Confirmed */
__constant__ static int Z_TIME[16] = { 82,83,84,85,86,87,88,243,249,250,251,252,253,254,255,253 }; /* Confirmed */
__constant__ static int Z_INGADJ[64] = { 18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33,34,35,36,37,38,39,40,41,42,43,23,24,31,32,33,34,61,62,63,64,65,66,67,68,69,70,71,72,73,74,75,76,77,78,79,80,81,82,83,84,85,86,87,88,89,90,91,92 };/* Confirmed */

__global__ void trigg(uint32_t threads, int *g_found, uint8_t *g_seed)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	uint8_t seed[16] = { 0 };
	uint32_t input[16], state[8];

	if (thread <= threads) {

		if (0 < thread <= 131071) { /* Total Permutations, this frame: 131,072 */
			seed[0] = Z_PREP[(thread & 7)];
			seed[1] = Z_TIMED[(thread >> 3) & 7];
			seed[2] = 1;
			seed[3] = 5;
			seed[4] = Z_NS[(thread >> 6) & 63];
			seed[5] = 1;
			seed[6] = Z_ING[(thread >> 12) & 31];
		}
		if (131071 < thread <= 262143) { /* Total Permutations, this frame: 131,072 */
			seed[0] = Z_TIME[(thread & 15)];
			seed[1] = Z_MASS[(thread >> 4) & 31];
			seed[2] = 1;
			seed[3] = Z_INF[(thread >> 9) & 15];
			seed[4] = 9;
			seed[5] = 2;
			seed[6] = 1;
			seed[7] = Z_AMB[(thread >> 13) & 15];
		}
		if (262143 < thread <= 4456447) { /* Total Permutations, this frame: 4,194,304 */
			seed[0] = Z_PREP[(thread & 7)];
			seed[1] = Z_TIMED[(thread >> 3) & 7];
			seed[2] = 1;
			seed[3] = Z_ADJ[(thread >> 6) & 63];
			seed[4] = Z_NPL[(thread >> 12) & 31];
			seed[5] = 1;
			seed[6] = Z_INGINF[(thread >> 17) & 31];
		}
		if (4456447 < thread <= 12845055) { /* Total Permutations, this frame: 8,388,608 */
			seed[0] = 5;
			seed[1] = Z_NS[(thread & 63)];
			seed[2] = 1;
			seed[3] = Z_PREP[(thread >> 6) & 7];
			seed[4] = Z_TIMED[(thread >> 9) & 7];
			seed[5] = Z_MASS[(thread >> 12) & 31];
			seed[6] = 3;
			seed[7] = 1;
			seed[8] = Z_ADJ[(thread >> 17) & 63];
		}
		if (12845055 < thread <= 29622271) { /* Total Permutations, this frame: 16,777,216 */
			seed[0] = Z_PREP[thread & 7];
			seed[1] = Z_ADJ[(thread >> 3) & 63];
			seed[2] = Z_MASS[(thread >> 9) & 31];
			seed[3] = 1;
			seed[4] = Z_NPL[(thread >> 14) & 31];
			seed[5] = 1;
			seed[6] = Z_INGINF[(thread >> 19) & 31];
		}
		if (29622271 < thread <= 46399487) { /* Total Permutations, this frame: 16,777,216 */
			seed[0] = Z_PREP[(thread & 7)];
			seed[1] = Z_MASS[(thread >> 3) & 31];
			seed[2] = 1;
			seed[3] = Z_ADJ[(thread >> 8) & 63];
			seed[4] = Z_NPL[(thread >> 14) & 31];
			seed[5] = 1;
			seed[6] = Z_INGINF[(thread >> 19) & 31];
		}
		if (46399487 < thread <= 63176703) { /* Total Permutations, this frame: 16,777,216 */
			seed[0] = Z_TIME[(thread & 15)];
			seed[1] = Z_AMB[(thread >> 4) & 15];
			seed[2] = 1;
			seed[3] = Z_ADJ[(thread >> 8) & 63];
			seed[4] = Z_MASS[(thread >> 14) & 31];
			seed[5] = 1;
			seed[6] = Z_ING[(thread >> 19) & 31];
		}
		if (63176703 < thread <= 600047615) { /* Total Permutations, this frame: 536,870,912 */
			seed[0] = Z_TIME[(thread & 15)];
			seed[1] = Z_AMB[(thread >> 4) & 15];
			seed[2] = 1;
			seed[3] = Z_PREP[(thread >> 8) & 7];
			seed[4] = 5;
			seed[5] = Z_ADJ[(thread >> 11) & 63];
			seed[6] = Z_NS[(thread >> 17) & 63];
			seed[7] = 3;
			seed[8] = 1;
			seed[9] = Z_INGADJ[(thread >> 23) & 63];
		}
		/* Below Two Frames are Valid, But Require 64-Bit Math: if extra entropy req'd.
		   if( < thread <= ) { /* Total Permutations, this frame: 549,755,813,888
			seed[ 0] = Z_ING[(thread & 31)];
			seed[ 1] = Z_PREP[(thread << 5) & 7];
			seed[ 2] = Z_TIME[(thread << 8) & 15];
			seed[ 3] = Z_MASS[(thread << 12) & 31];
			seed[ 4] = 1;
				seed[ 5] = Z_MASS[(thread << 17) & 31];
			seed[ 6] = Z_ING[(thread << 22) & 31];
			seed[ 7] = 3;
			seed[ 8] = 1;
				seed[ 9] = 5;
			seed[10] = Z_ADJ[(thread << 27) & 63];
			seed[11] = Z_NS[(thread << 33) & 63];
		   }
		   if( < thread <= ) { /* Total Permutations, this frame: 4,398,046,511,104
			seed[ 0] = Z_ING[(thread & 31)];
			seed[ 1] = Z_PREP[(thread << 5) & 7];
			seed[ 2] = 5;
			seed[ 3] = Z_ADJ[(thread << 8) & 63];
			seed[ 4] = Z_NS[(thread << 14) & 63];
			seed[ 5] = 1;
				seed[ 6] = Z_MASS[(thread << 19) & 31];
			seed[ 7] = Z_ING[(thread << 24) & 31];
			seed[ 8] = 3;
			seed[ 9] = 1;
				seed[10] = 5;
			seed[11] = Z_ADJ[(thread << 30) & 63];
			seed[12] = Z_NS[(thread << 36) & 63];
		   }
		End 64-bit Frames */

#pragma unroll
		for (int i = 0; i < 8; i++)
		{
			input[i] = c_input32[i];
		}
#pragma unroll
		for (int i = 0; i < 4; i++)
		{
			input[8 + i] = cuda_swab32(((uint32_t *)seed)[i]);
		}

		input[12] = cuda_swab32(c_blockNumber8[0]);
		input[13] = cuda_swab32(c_blockNumber8[1]);
		input[14] = 0x80000000;
		input[15] = 0;

#pragma unroll
		for (int i = 0; i < 8; i += 2)
		{
			AS_UINT2(&state[i]) = AS_UINT2(&c_midstate256[i]);
		}

		sha256_round(input, state, c_K);

#pragma unroll
		for (int i = 0; i < 15; i++)
		{
			input[i] = 0;
		}
		input[15] = 0x9c0;

		sha256_round(input, state, c_K);

		if (gpu_trigg_eval(state, c_difficulty))
		{
			*g_found = 1;
#pragma unroll
			for (int i = 0; i < 16; i++)
			{
				g_seed[i] = seed[i];
			}
		}
	}
}

typedef struct __trigg_cuda_ctx {
	byte curr_seed[16], next_seed[16];
	char cp[256], *next_cp;
	int *found, *d_found;
	uint8_t *seed, *d_seed;
	uint32_t *midstate, *input;
} TriggCudaCTX;

/* Max 64 GPUs Supported */
TriggCudaCTX ctx[64];
int thrds = 600;
int threads = 600047615;
dim3 grid(585984);
dim3 block(1024);
char *nullcp = '\0';
byte *diff;
byte *bnum;
int nGPU = 0;
hipStream_t streams[64];

__host__ int count_devices_cuda() {
	int num_devices;
	/* Obtain and check system GPU count */
	hipGetDeviceCount(&num_devices);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr,
			"Unable to count CUDA devices, error: (%d) %s.\n",
			static_cast<int>(err), hipGetErrorString(err));
		return 0;
	}
	return num_devices;
}

__host__ int trigg_init_cuda(byte difficulty, byte *blockNumber) {
	/* Obtain and check system GPU count */
	checkCudaErrors(hipGetDeviceCount(&nGPU));
	if (nGPU < 1 || nGPU>64) return nGPU;
	/* Allocate pinned host memory */
	checkCudaErrors(hipHostMalloc(&diff, 1));
	checkCudaErrors(hipHostMalloc(&bnum, 8));
	/* Copy immediate block data to pinned memory */
	memcpy(diff, &difficulty, 1);
	memcpy(bnum, blockNumber, 8);

	int i = 0;
	for (; i < nGPU; i++) {
		checkCudaErrors(hipSetDevice(i));
		/* Allocate device memory */
		checkCudaErrors(hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking));
		checkCudaErrors(hipMalloc(&ctx[i].d_found, 4));
		checkCudaErrors(hipMalloc(&ctx[i].d_seed, 16));
		/* Allocate associated device-host memory */
		checkCudaErrors(hipHostMalloc(&ctx[i].found, 4));
		checkCudaErrors(hipHostMalloc(&ctx[i].seed, 16));
		checkCudaErrors(hipHostMalloc(&ctx[i].midstate, 32));
		checkCudaErrors(hipHostMalloc(&ctx[i].input, 32));
		/* Copy immediate block data to device memory */
		checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_blockNumber8), bnum, 8, 0, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_difficulty), diff, 1, 0, hipMemcpyHostToDevice));
		/* Set remaining device memory */
		checkCudaErrors(hipMemsetAsync(ctx[i].d_found, 0, 4));
		checkCudaErrors(hipMemsetAsync(ctx[i].d_seed, 0, 16));
		/* Setup variables for "first round" */
		*ctx[i].found = 0;
		ctx[i].next_cp = nullcp;
		printf("\nTrace: GPU %d Initialized.", i);
	}

	return nGPU;
}

__host__ void trigg_free_cuda() {
	/* Free pinned host memory */
	checkCudaErrors(hipHostFree(diff));
	checkCudaErrors(hipHostFree(bnum));

	int i = 0;
	for (; i < nGPU; i++) {
		checkCudaErrors(hipSetDevice(i));
		checkCudaErrors(hipStreamDestroy(streams[i]));
		/* Free device memory */
		checkCudaErrors(hipFree(ctx[i].d_found));
		checkCudaErrors(hipFree(ctx[i].d_seed));
		/* Free associated device-host memory */
		checkCudaErrors(hipHostFree(ctx[i].found));
		checkCudaErrors(hipHostFree(ctx[i].seed));
		checkCudaErrors(hipHostFree(ctx[i].midstate));
		checkCudaErrors(hipHostFree(ctx[i].input));
		printf("\nTrace: GPU %d Freed.", i);

	}
}

__host__ char *trigg_generate_cuda(byte *mroot, uint32_t *nHaiku)
{
	int i;

	for (i = 0; i < nGPU; i++) {
		/*printf("\nFound value is:  %d", *ctx[i].found);*/
		/* If next_cp is empty... */
		if (ctx[i].next_cp == nullcp) {
			/* ... init GPU seeds */
			trigg_gen(ctx[i].next_seed);
			ctx[i].next_cp = trigg_expand(ctx[i].next_seed, *diff);
			/* ... copy mroot to Tchain */
			memcpy(Tchain, mroot, 32);
			/* ... and prepare sha256 midstate for next round */
			SHA256_CTX sha256;
			sha256_init(&sha256);
			sha256_update(&sha256, Tchain, 256);
			memcpy(ctx[i].midstate, sha256.state, 32);
			memcpy(ctx[i].input, Tchain + 256, 32);
		}

		if (hipStreamQuery(streams[i]) == hipSuccess) hipMemcpy(ctx[i].found, ctx[i].d_found, 4, hipMemcpyDeviceToHost);
		/** Due to the asynchronous nature of this process,
		 ** conditions below MUST be performed in order of
		 ** found status (-1) to (1), so a solve isn't "missed" **/

		 /* Waiting on GPU || *ctx[i].found == -1 */
		if (*ctx[i].found < 0) continue;

		/* GPU is done. NO SOLVE || *ctx[i].found == 0 From First Run */
		if (*ctx[i].found < 1) {
			/* Start new GPU round */
			checkCudaErrors(hipSetDevice(i));
			checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_midstate256), ctx[i].midstate, 32, 0, hipMemcpyHostToDevice));
			checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_input32), ctx[i].input, 32, 0, hipMemcpyHostToDevice));
			trigg << <grid, block, 0, streams[i] >> > (threads, ctx[i].d_found, ctx[i].d_seed);

			/* Set GPU waiting status and add to haiku count */
			*nHaiku += thrds;
			*ctx[i].found = -1;

			/* Store round vars aside for checks next loop */
			memcpy(ctx[i].curr_seed, ctx[i].next_seed, 16);
			strcpy(ctx[i].cp, ctx[i].next_cp);
			ctx[i].next_cp = nullcp;
			continue;
		}

		/* GPU is done. SOLVED! || *ctx[i].found == 1 */
		checkCudaErrors(hipSetDevice(i));
		checkCudaErrors(hipMemcpy(ctx[i].seed, ctx[i].d_seed, 16, hipMemcpyDeviceToHost));
		memcpy(mroot + 32, ctx[i].curr_seed, 16);
		memcpy(mroot + 32 + 16, ctx[i].seed, 16);
		return ctx[i].cp;
	}

	return NULL;
}