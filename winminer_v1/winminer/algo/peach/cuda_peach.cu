#include "hip/hip_runtime.h"
/*
 * cuda_trigg.cu  Multi-GPU CUDA Mining
 *
 * Copyright (c) 2019 by Adequate Systems, LLC.  All Rights Reserved.
 * See LICENSE.PDF   **** NO WARRANTY ****
 *
 * Date: 10 August 2018
 * Revision: 31
 */

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "../../sleep.h"
#include "../../config.h"

#include "../../types.h"

#include "peach.h"
#include "nighthash.cu"

byte *trigg_gen(byte *in);

__constant__ static uint8_t __align__(8) c_phash[32];
__constant__ static uint8_t __align__(8) c_input[108];
__constant__ static uint8_t __align__(8) c_difficulty;
__constant__ static int Z_MASS[4] = {238,239,240,242};
__constant__ static int Z_ING[2]  = {42,43};
__constant__ static int Z_TIME[16] =
   {82,83,84,85,86,87,88,243,249,250,251,252,253,254,255,253};
__constant__ static int Z_AMB[16] =
   {77,94,95,96,126,214,217,218,220,222,223,224,225,226,227,228};
__constant__ static int Z_ADJ[64] =
   {61,62,63,64,65,66,67,68,69,70,71,72,73,74,75,76,77,78,79,80,81,82,83,84,85,86,87,
    88,89,90,91,92,94,95,96,97,98,99,100,101,102,103,104,105,107,108,109,110,112,114,
    115,116,117,118,119,120,121,122,123,124,125,126,127,128};

inline int cudaCheckError( const char *msg, uint32_t gpu, const char *file)
{
   hipError_t err = hipGetLastError();
   if(hipSuccess != err) {
      fprintf(stderr, "%s Error (#%d) in %s: %s\n",
              msg, gpu, file, hipGetErrorString(err));
      return 1;
   }
   return 0;
}


__device__ uint32_t cuda_next_index(uint32_t index, uint8_t *g_map, uint8_t *nonce, uint8_t debug)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[HASHLEN + 4 + TILE_LENGTH];
   byte hash[HASHLEN];
   int i, seedlen;

   /* Create nighthash seed for this index on the map */
   seedlen = HASHLEN + 4 + TILE_LENGTH;
   memcpy(seed, nonce, HASHLEN);
   memcpy(seed + HASHLEN, (byte *) &index, 4);
   memcpy(seed + HASHLEN + 4, &g_map[index * TILE_LENGTH], TILE_LENGTH);

   if (debug) {
	   printf("first tile: ");
	   for (int i = 0; i < TILE_LENGTH; i++) {
		   printf("%02x ", g_map[index * TILE_LENGTH + i]);
	   }
	   printf("\n");

	   printf("cuda_next_index seed: ");
	   for (int i = 0; i < seedlen; i++) {
		   printf("%02x ", seed[i]);
	   }
	   printf("\n");
   }
   
   /* Setup nighthash the seed, NO TRANSFORM */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 0, debug);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen, debug);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, hash, debug);

   /* Convert 32-byte Hash Value Into 8x 32-bit Unsigned Integer */
   for(i = 0, index = 0; i < 8; i++)
      index += ((uint32_t *) hash)[i];

   return index % MAP;
}


__device__ void cuda_gen_tile(uint32_t index, uint8_t *g_map, uint8_t debug)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[4 + HASHLEN];
   byte *tilep;
   int i, j, seedlen;

   /* Set map pointer */
   tilep = &g_map[index * TILE_LENGTH];

   /* Create nighthash seed for this index on the map */
   seedlen = 4 + HASHLEN;
   memcpy(seed, (byte *) &index, 4);
   memcpy(seed + 4, c_phash, HASHLEN);
  
   /* Setup nighthash with a transform of the seed */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 1, debug);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen, debug);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, tilep, debug);

   /* Begin constructing the full tile */
   for(i = 0; i < TILE_LENGTH; i += HASHLEN) { /* For each tile row */
      /* Set next row's pointer location */
      j = i + HASHLEN;

      /* Hash the current row to the next, if not at the end */
      if(j < TILE_LENGTH) {
         /* Setup nighthash with a transform of the current row */
         cuda_nighthash_init(&nighthash, &tilep[i], HASHLEN, index, 1, debug);

         /* Update nighthash with the seed data and tile index */
         cuda_nighthash_update(&nighthash, &tilep[i], HASHLEN, debug);
         cuda_nighthash_update(&nighthash, (byte *) &index, 4, debug);

         /* Finalize nighthash into the first 32 byte chunk of the tile */
         cuda_nighthash_final(&nighthash, &tilep[j], debug);

		 if (debug) {
			 printf("tile_part[%d] = ", i / HASHLEN);
			 for (int z = 0; z < HASHLEN; z++) {
				 printf("%02x ", tilep[i + z]);
			 }
			 printf("\n");
			 printf("tile_part[%d] = ", j / HASHLEN);
			 for (int z = 0; z < HASHLEN; z++) {
				 printf("%02x ", tilep[j+z]);
			 }
			 printf("\n");
		 }
      }
   }
}


__global__ void cuda_build_map(uint8_t *g_map)
{
   const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
   if (thread < MAP)
      cuda_gen_tile(thread, g_map, /*thread == 0 ? 1 : 0*/ 0);

   /*if (thread == 0) {
	   printf("tile 0: ");
	   for (int i = 0; i < TILE_LENGTH; i++) {
		   printf("%02x ", g_map[i]);
	   }
	   printf("\n");
   }*/
}


__global__ void cuda_find_peach(uint32_t threads, uint8_t *g_map,
                                int32_t *g_found, uint8_t *g_seed)
{
   const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

   CUDA_SHA256_CTX ictx;
   uint8_t seed[16] = {0}, nonce[32] = {0};
   uint8_t bt_hash[32], fhash[32];
   int32_t i, j, n, x;
   uint32_t sm;

   if (thread < threads) {
      /* Determine second seed */
      if(thread < 131072) { /* This frame permutations: 131,072 */
         seed[ 0] = Z_TIME[(thread & 15)];
         seed[ 1] = Z_AMB[(thread >> 4) & 15];
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];
         seed[ 4] = Z_MASS[(thread >> 14) & 3];
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 16) & 1];
      }

      /* store full nonce */
      #pragma unroll
      for (i = 0; i < 16; i++)
         nonce[i] = c_input[i + 92];

      #pragma unroll
      for (i = 0; i < 16; i++)
         nonce[i+16] = seed[i];

      /*********************************************************/
      /* Hash 124 bytes of Block Trailer, including both seeds */

      cuda_sha256_init(&ictx);
      cuda_sha256_update(&ictx, c_input, 108);
      cuda_sha256_update(&ictx, seed, 16);
      cuda_sha256_final(&ictx, bt_hash);

      /****************************************************/
      /* Follow the tile path based on the selected nonce */
      
      sm = bt_hash[0];
      #pragma unroll
      for(i = 1; i < HASHLEN; i++)
         sm *= bt_hash[i];
      sm %= MAP;

      /* make <JUMP> tile jumps to find the final tile */
      #pragma unroll
      for(j = 0; j < JUMP; j++)
        sm = cuda_next_index(sm, g_map, nonce, 0);

      /****************************************************************/
      /* Check the hash of the final tile produces the desired result */

      cuda_sha256_init(&ictx);
      cuda_sha256_update(&ictx, bt_hash, HASHLEN);
      cuda_sha256_update(&ictx, &g_map[sm * TILE_LENGTH], TILE_LENGTH);
      cuda_sha256_final(&ictx, fhash);

      /* Evaluate hash */
      for (x = i = j = n = 0; i < HASHLEN; i++) {
         x = fhash[i];
         if (x != 0) {
            for(j = 7; j > 0; j--) {
               x >>= 1;
               if(x == 0) {
                  n += j;
                  break;
               }
            }
            break;
         }
         n += 8;
      }

	  if (n >= c_difficulty && !atomicExch(g_found, 1)) {
		  /* PRINCESS FOUND! */
         #pragma unroll
         for (i = 0; i < 16; i++)
            g_seed[i] = seed[i];
      }
      /* Our princess is in another castle ! */
   }
   else {
	   printf("WARNING: thread >= threads: %d\n", thread);
   }
}


extern "C" {

typedef struct __peach_cuda_ctx {
   byte init, curr_seed[16], next_seed[16];
   byte *seed, *d_seed;
   byte *input, *d_map;
   int32_t *d_found;
   hipStream_t stream;
} PeachCudaCTX;

/* Max 63 GPUs Supported */
PeachCudaCTX ctx[64];
dim3 grid(512);
dim3 block(256);
uint32_t threads = 131072;
int32_t nGPU = 0;
int32_t *found;
byte gpuInit = 0;
byte bnum[8] = {0};
byte *diff;
byte *phash;

int init_cuda_peach(byte difficulty, byte *prevhash, byte *blocknumber) {
	printf("init_cuda_peach\n");
   int i;
   
   /* Obtain and check system GPU count */
   nGPU = 0;
   hipGetDeviceCount(&nGPU);
   if(nGPU<1 || nGPU>64) return nGPU;
   
   /* Allocate pinned host memory */
   hipHostMalloc(&found, 4);
   if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
	   printf("hipHostMalloc Failed\n");
   }
   hipHostMalloc(&diff, 1);
   if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
	   printf("hipHostMalloc Failed\n");
   }
   hipHostMalloc(&phash, 32);
   if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
	   printf("hipHostMalloc Failed\n");
   }
   
   /* Copy immediate block data to pinned memory */
   *found = 0;
   *diff = difficulty;
   memcpy(phash, prevhash, 32);
   
   /* Initialize GPU context init variable incase
    * it holds a random number from memory */
   if(gpuInit == 0) {
      gpuInit = 1;
      for (i = 0; i < nGPU; i++)
         ctx[i].init = 0;
   }
   
   printf("Found %d GPUs\n", nGPU);
   /* Initialize GPU data asynchronously */
   for (i = 0; i < nGPU; i++) {
      hipSetDevice(i);
      
      /* Create Stream */
      hipStreamCreate(&ctx[i].stream);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipStreamCreate failed\n");
		  return -1;
	  }
      
      /* Allocate device memory */
      hipMalloc(&ctx[i].d_found, 4);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipMalloc failed\n");
		  return -1;
	  }
      hipMalloc(&ctx[i].d_seed, 16);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipMalloc failed\n");
		  return -1;
	  }
      
      /* Allocate associated device-host memory */
      hipHostMalloc(&ctx[i].seed, 16);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipHostMalloc failed\n");
		  return -1;
	  }
      hipHostMalloc(&ctx[i].input, 108);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipHostMalloc failed\n");
		  return -1;
	  }
      
      /* Copy immediate block data to device memory */
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_difficulty), diff, 1, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipMemcpyToSymbolAsync failed\n");
		  return -1;
	  }
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_phash), phash, 32, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipMemcpyToSymbolAsync failed\n");
		  return -1;
	  }
      
      /* Set remaining device memory */
      hipMemsetAsync(ctx[i].d_found, 0, 4, ctx[i].stream);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipMemsetAsync failed\n");
		  return -1;
	  }
      hipMemsetAsync(ctx[i].d_seed, 0, 16, ctx[i].stream);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipMemsetAsync failed\n");
		  return -1;
	  }
      
      /* Set initial round variables */
      ctx[i].next_seed[0] = 0;
      
      /* If first init, setup map and cache */
      if(ctx[i].init == 0) {
         /* NOTE: The device MAP that holds the data of a map DOES NOT
          * explicitly get free()'d. The reason behind this is because
          * we reuse the map variable between blocks, and just rebuild
          * the map once every block. The GPU free's the MAP when the
          * program ends by default. This can be adjusted later. */
         hipMalloc(&ctx[i].d_map, MAP_LENGTH);
		 if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
			 printf("hipMalloc failed\n");
			 return -1;
		 }
         ctx[i].init = 1;
      }
      
      /* (re)Build map if new block */
	  if (memcmp(bnum, blocknumber, 8) != 0) {
		  printf("Rebuilding map\n");
		  hipStreamSynchronize(ctx[i].stream);
		  hipDeviceSynchronize();
		  cuda_build_map << <4096, 256, 0, ctx[i].stream >> > (ctx[i].d_map);

		  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
			  printf("Problem with cuda_build_map\n");
			  return -1;
		  }
	  }
   }
   
   /* Check for any GPU initialization errors */
   for(i = 0; i < nGPU; i++) {
      hipSetDevice(i);
	  printf("Wait for synchronization of cuda_build_map\n");
      hipStreamSynchronize(ctx[i].stream);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("error in cuda\n");
		  return -1;
	  }
   }
   
   /* Update block number */
   memcpy(bnum, blocknumber, 8);

   printf("Returning %d gpus\n", nGPU);
   return nGPU;
}

void free_cuda_peach() {
	printf("free_cuda_peach\n");
   int i;
   
   /* Free pinned host memory */
   hipHostFree(diff);
   hipHostFree(found);
   hipHostFree(phash);
   
   /* Free GPU data */
   for (i = 0; i<nGPU; i++) {
      hipSetDevice(i);
      
      /* Destroy Stream */
      hipStreamDestroy(ctx[i].stream);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipStreamDestroy failed\n");
	  }
      
      /* Free device memory */
      hipFree(ctx[i].d_found);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipFree failed\n");
	  }
      hipFree(ctx[i].d_seed);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipFree failed\n");
	  }
      /*hipFree(ctx[i].d_map);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipFree failed\n");
	  }*/
      
      /* Free associated device-host memory */
      hipHostFree(ctx[i].seed);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipHostFree failed\n");
	  }
      hipHostFree(ctx[i].input);
	  if (cudaCheckError("init_cuda_peach()", i, __FILE__)) {
		  printf("hipHostFree failed\n");
	  }
   }
}


__host__ void cuda_peach(byte *bt, uint32_t *hps, byte *runflag)
{
	printf("cuda_peach\n");
   int i;
   uint64_t lastnHaiku, nHaiku = 0;
   time_t seconds = time(NULL);
   for( ; *runflag && *found == 0; ) {
      for (i=0; i<nGPU; i++) {
         /* Prepare next seed for GPU... */
         if(ctx[i].next_seed[0] == 0) {
            /* ... generate first GPU seed (and expand as Haiku) */
            trigg_gen(ctx[i].next_seed);

            /* ... and prepare round data */
            memcpy(ctx[i].input, bt, 92);
            memcpy(ctx[i].input+92, ctx[i].next_seed, 16);
         }
         /* Check if GPU has finished */
         hipSetDevice(i);
         if(hipStreamQuery(ctx[i].stream) == hipSuccess) {
			 printf("gpu completed, threads: %d\n", threads);
            hipMemcpy(found, ctx[i].d_found, 4, hipMemcpyDeviceToHost);
            if(*found==1) { /* SOLVED A BLOCK! */
               hipMemcpy(ctx[i].seed, ctx[i].d_seed, 16, hipMemcpyDeviceToHost);
               memcpy(bt + 92, ctx[i].curr_seed, 16);
               memcpy(bt + 92 + 16, ctx[i].seed, 16);
               break;
            }
            /* Send new GPU round Data */
            hipMemcpyToSymbolAsync(HIP_SYMBOL(c_input), ctx[i].input, 108, 0,
                                    hipMemcpyHostToDevice, ctx[i].stream);
			hipStreamSynchronize(ctx[i].stream);
            /* Start GPU round */
			printf("starting new gpu round\n");
            cuda_find_peach<<<grid, block, 0, ctx[i].stream>>>(threads,
                                 ctx[i].d_map, ctx[i].d_found, ctx[i].d_seed);
			if (cudaCheckError("cuda_peach()", i, __FILE__)) {
				printf("CUDA ERROR\n");
			}

            /* Add to haiku count */
            nHaiku += threads;

            /* Store round vars aside for checks next loop */
            memcpy(ctx[i].curr_seed,ctx[i].next_seed,16);
			
            ctx[i].next_seed[0] = 0;
         }
         
         /* Waiting on GPU? ... */
         if(cudaCheckError("cuda_peach()", i, __FILE__)) {
            *runflag = 0;
            return;
         }
      }
      
      /* Chill a bit if nothing is happening */
      if(lastnHaiku == nHaiku) msleep(1);
      else lastnHaiku = nHaiku;
   }
    
   seconds = time(NULL) - seconds;
   if(seconds == 0) seconds = 1;
   nHaiku /= seconds;
   *hps = (uint32_t) nHaiku;
}


}
